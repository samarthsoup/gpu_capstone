
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>   
#include <ctime>     
#include <sstream>   

int main(int argc, char* argv[]) 
{
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <number of random numbers>\n";
        return 1;
    }

    int n = 0;
    std::istringstream iss(argv[1]);
    if (!(iss >> n) || n <= 0) {
        std::cerr << "Invalid number of random numbers: " << argv[1] << "\n";
        return 1;
    }


    std::srand(static_cast<unsigned int>(std::time(0)));

    std::ofstream outfile("data/generated_data.txt");
    if (!outfile) {
        std::cerr << "Error: Could not open output file.\n";
        return 1;
    }

    outfile << n << " ";

    for (int i = 0; i < n; ++i) {
        int random_number = std::rand() % 100; 
        outfile << random_number << " ";  
    }

    outfile.close();  // Close the file
    std::cout << "Successfully wrote " << n << " random numbers to 'generated_data.txt'\n";

    return 0;
}
