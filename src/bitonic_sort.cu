#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <iostream>
#include <fstream>

#define MAX_THREADS_PER_BLOCK 1024

void bitonicSortCPU(int* arr, int n) 
{
    for (int k = 2; k <= n; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            for (int i = 0; i < n; i++) {
                int ij = i ^ j;

                if (ij > i) {
                    if ((i & k) == 0) {
                        if (arr[i] > arr[ij]){
                            int temp = arr[i];
                            arr[i] = arr[ij];
                            arr[ij] = temp;
                        }
                    } else {
                        if (arr[i] < arr[ij]){
                            int temp = arr[i];
                            arr[i] = arr[ij];
                            arr[ij] = temp;
                        }
                    }
                }
            }
        }
    }
}

__global__ void bitonicSortGPU(int* arr, int j, int k)
{
    unsigned int i, ij;

    i = threadIdx.x + blockDim.x * blockIdx.x;

    ij = i ^ j;

    if (ij > i) {
        if ((i & k) == 0) {
            if (arr[i] > arr[ij]) {
                int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }
        } else {
            if (arr[i] < arr[ij]) {
                int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }
        }
    }
}

void printArray(int* arr, int size) 
{
    for (int i = 0; i < size; ++i)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

bool isSorted(int* arr, int size) 
{
    for (int i = 1; i < size; ++i) 
    {
        if (arr[i] < arr[i - 1])
            return false;
    }
    return true;
}

bool isPowerOfTwo(int num) 
{
    return num > 0 && (num & (num - 1)) == 0;
}

int nextPowerOfTwo(int n) {
    if (n && !(n & (n - 1))) {
        return n;
    }
    
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;

    return n;
}


int main() 
{   
    std::ifstream infile("data/generated_data.txt");
    if (!infile) {
        std::cerr << "Error opening file.\n";
        return 1;
    }

    int input_size = 0;
    infile >> input_size;

    int size;
    if (input_size <= 0) {
        std::cerr << "Array size must be a positive integer\n";
        return 1;
    }

    if (!isPowerOfTwo(input_size)) {   
        std::cout << "Size provided is not a power of two, size will be the next power of two and remaining spots of the array will be padded with zeroes\nSize provided: " << input_size << std::endl;
        size = nextPowerOfTwo(input_size);
        std::cout << "The nearest higher power of two is: " << size << std::endl;
    } else {
        size = input_size;
    }

    int* arr = new int[size];
    int* carr = new int[size];
    int* temp = new int[size];

    int* gpuArrbiton;
    int* gpuTemp;

    srand(static_cast<unsigned int>(time(nullptr)));
    for (int i = 0; i < input_size; ++i) {
        if (!(infile >> arr[i])) {
            std::cerr << "Error reading number at position " << i + 1 << ".\n";
            delete[] arr;
            return 1;
        }
        carr[i] = arr[i];
    }

    infile.close();

    for (int i = input_size; i < size; ++i) {
        arr[i] = 0;
        carr[i] = 0;
    }

    hipMalloc((void**)&gpuTemp, size * sizeof(int));
    hipMalloc((void**)&gpuArrbiton, size * sizeof(int));

    hipMemcpy(gpuArrbiton, arr, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    float GPU_time_ms = 0;

    clock_t startCPU, endCPU;

    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    
    int j, k;

    hipEventRecord(startGPU);
    for (k = 2; k <= size; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonicSortGPU << <blocksPerGrid, threadsPerBlock >> > (gpuArrbiton, j, k);
        }
    }
    hipEventRecord(stopGPU);

    hipMemcpy(arr, gpuArrbiton, size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stopGPU);
    hipEventElapsedTime(&GPU_time_ms, startGPU, stopGPU);

    startCPU = clock();
    bitonicSortCPU(carr, size);
    endCPU = clock();
    

    double CPU_time_ms = static_cast<double>(endCPU - startCPU) / (CLOCKS_PER_SEC / 1000.0);
    
    if (isSorted(arr, size))
        std::cout << "\n\nSORT CHECKER RUNNING - SUCCESFULLY SORTED GPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;
   
    if (isSorted(carr, size))
        std::cout << "SORT CHECKER RUNNING - SUCCESFULLY SORTED CPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;

    std::cout << "\n\nGPU Time: " << GPU_time_ms << " ms" << std::endl;
    std::cout << "CPU Time: " << CPU_time_ms << " ms" << std::endl;

    std::ofstream outfile("data/output.txt");
    if (!outfile) {
        std::cerr << "Error opening output file.\n";
        delete[] arr;
        return 1;
    }

    for (int i = size-input_size; i < size; ++i) {
        outfile << arr[i] << " ";
    }

    outfile.close();

    delete[] arr;
    delete[] carr;
    delete[] temp;

    hipFree(gpuArrbiton);
    hipFree(gpuTemp);

    return 0;
}